#include "cuda.cu.h"

#include <cstdio>
#include <ctime>
#include <stdio.h>
#include <stdint.h>

#include "handle_error.cu.h"
//#include "netmodel/cmodel_sorter.cu.h"
#include "netmodel/cmodel_gan.cu.h"
//#include "netmodel/cmodel_srgan.cu.h"
//#include "netmodel/cmodel_vae.cu.h"
#include "../system/system.h"

//----------------------------------------------------------------------------------------------------
//главная функция программы на CUDA
//----------------------------------------------------------------------------------------------------

void CUDA_Start(void)
{
/*
struct hipDeviceProp_t
{
    char   name[256];
    size_t totalGlobalMem;
    size_t sharedMemPerBlock;
    int    regsPerBlock;
    int    warpSize;
    size_t memPitch;
    int    maxThreadsPerBlock;
    int    maxThreadsDim [3];
    int    maxGridSize   [3];
    size_t totalConstMem;
    int    major;
    int    minor;
    int    clockRate;
    size_t textureAlignment;
    int    deviceOverlap;
    int    multiProcessorCount;
}
*/


 int deviceCount;
 hipDeviceProp_t devProp;

 HANDLE_ERROR(hipGetDeviceCount(&deviceCount));
 printf("Found %d devices\n",deviceCount);
 for(int device=0;device<deviceCount;device++)
 {
  char str[1024];
  HANDLE_ERROR(hipGetDeviceProperties(&devProp,device));
  sprintf(str,"Device %d\n", device );
  SYSTEM::PutMessageToConsole(str);
  sprintf(str,"Compute capability     : %lu.%lu",static_cast<long unsigned int>(devProp.major),static_cast<long unsigned int>(devProp.minor));
  SYSTEM::PutMessageToConsole(str);
  sprintf(str,"Name                   : %s",devProp.name);
  SYSTEM::PutMessageToConsole(str);
  sprintf(str,"Total Global Memory    : %lu",static_cast<long unsigned int>(devProp.totalGlobalMem));
  SYSTEM::PutMessageToConsole(str);
  sprintf(str,"Shared memory per block: %lu",static_cast<long unsigned int>(devProp.sharedMemPerBlock));
  SYSTEM::PutMessageToConsole(str);
  sprintf(str,"Registers per block    : %lu",static_cast<long unsigned int>(devProp.regsPerBlock));
  SYSTEM::PutMessageToConsole(str);
  sprintf(str,"Warp size              : %lu",static_cast<long unsigned int>(devProp.warpSize));
  SYSTEM::PutMessageToConsole(str);
  sprintf(str,"Max threads per block  : %lu",static_cast<long unsigned int>(devProp.maxThreadsPerBlock));
  SYSTEM::PutMessageToConsole(str);

  sprintf(str,"Max Grid Size: %lux%lux%lu",static_cast<long unsigned int>(devProp.maxGridSize[0]),static_cast<long unsigned int>(devProp.maxGridSize[1]),static_cast<long unsigned int>(devProp.maxGridSize[2]));
  SYSTEM::PutMessageToConsole(str);

  sprintf(str,"Max Threads dim: %lux%lux%lu",static_cast<long unsigned int>(devProp.maxThreadsDim[0]),static_cast<long unsigned int>(devProp.maxThreadsDim[1]),static_cast<long unsigned int>(devProp.maxThreadsDim[2]));
  SYSTEM::PutMessageToConsole(str);

  sprintf(str,"Total constant memory  : %lu",devProp.totalConstMem);
  SYSTEM::PutMessageToConsole(str);
 }
 HANDLE_ERROR(hipSetDevice(0));
 HANDLE_ERROR(hipGetLastError());
 HANDLE_ERROR(hipDeviceReset());
 HANDLE_ERROR(hipGetLastError());

 if (CTensorTest<float>::Test()==false) throw("Класс тензоров провалил тестирование!");


 //CModelSorter<float> cModelSorter;
 //cModelSorter.Execute();

 CModelGAN<float> cModelGAN;
 cModelGAN.Execute();

 //CModelVAE<float> cModelVAE;
 //cModelVAE.Execute();
/*
 CModelSR_GAN<float> cModelSR_GAN;
 cModelSR_GAN.Execute();
 */
}
